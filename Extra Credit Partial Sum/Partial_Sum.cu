#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "Partial_Sum_Kernel.cu"

#define BLOCK_SIZE  32;
#define SAMPLE_SIZE 32

void FATAL (const char * s )
{
	puts(s);
	exit(1);
}

int main(int argc, char**argv) {
    
    unsigned int data_size;
    hipError_t cuda_ret;

    
    if(argc == 1) {
        data_size= 64;
    } else if(argc == 2) {
        data_size= atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

//Device data
int * array_dev;
int array_size = data_size;

//Host data
int * array_host = (int *) malloc (sizeof(int)*array_size);
for(int i = 0; i < data_size; ++i)
	array_host[i] = i + 1;

for (int i = data_size; i < array_size; ++i)
	array_host[i] = 0;

int expected_sum = data_size * (array_host[0] + array_host[data_size - 1]) / 2;

//Allocating & copying device memory
cuda_ret = hipMalloc((void**)&array_dev, array_size*sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
cuda_ret = hipMemcpy(array_dev, array_host, array_size*sizeof(int), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

hipDeviceSynchronize();

//Invoke Kernel}
vecSum_final_int1<<<dim3(1, 1, 1), dim3(SAMPLE_SIZE, 1, 1)>>>(array_dev);
vecSum_final_int<<<dim3(1, 1, 1), dim3(SAMPLE_SIZE, 1, 1)>>>(array_dev);

//Copying to host memory
int *result = (int *) malloc(sizeof(int)*array_size);
cuda_ret = hipMemcpy(result, array_dev, sizeof(int)*array_size, hipMemcpyDeviceToHost);
if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

hipDeviceSynchronize();

printf("Array size = %d\n", array_size);
printf("Expected result = %d\n", expected_sum);
printf("Calculated result = %d\n", result[0]);

for (int i = 0; i < data_size; ++i){
	printf("[%2d] : %5d, %5d\n", i, array_host[i], result[i]);
}


fflush(stdout);

free(array_host);
hipFree(array_dev);

return 0;

};
