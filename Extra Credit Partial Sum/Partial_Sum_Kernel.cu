
#include <hip/hip_runtime.h>

__global__ void vecSum_final_int(int * array)
{


  for(unsigned int offset = blockDim.x; offset  > 0; offset = offset >> 1){
      __syncthreads();

      if (threadIdx.x < offset)
          array[threadIdx.x] += array[threadIdx.x + offset];
  }
}

__global__ void vecSum_final_int1(int * array)
{
  const int tidx = threadIdx.x << 1;

  for (unsigned int stride = 1; stride <= blockDim.x; stride = stride << 1 ){

      __syncthreads();

      if(tidx % stride == 0)
         array[tidx] += array[tidx + stride];

 }
}
