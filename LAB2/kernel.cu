/**********:{********************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int row, col;

    row = blockIdx.y*blockDim.y+threadIdx.y;
    
    col = blockIdx.x*blockDim.x+threadIdx.x;
    
   
    if(( row < m) && (col < n))
    {
	float acc = 0;

	for(int index = 0; index < k; index++)
        {
	   acc = acc + A[row * k + index] * B[index * n + col];
        }

        C[row * n + col] = acc;  			

    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE

    dim3 block(BLOCK_SIZE, BLOCK_SIZE ,1);
    dim3 grid((n + BLOCK_SIZE - 1)/BLOCK_SIZE, (m + BLOCK_SIZE -1)/BLOCK_SIZE, 1);
    

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE

    mysgemm<<< grid, block>>>(m, n, k, A, B, C);

}
