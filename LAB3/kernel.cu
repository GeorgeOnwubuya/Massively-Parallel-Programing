/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SZ 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    unsigned int TiRow = threadIdx.y;
    unsigned int TiCol = threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float As[TILE_SZ][TILE_SZ];
    __shared__ float Bs[TILE_SZ][TILE_SZ];

    float sum = 0;


    for(unsigned int TiNum = 0; TiNum < (k-1)/TILE_SZ+1; TiNum++){
  	if((row < m) && (TiNum * TILE_SZ + TiCol) < k)
    		As[TiRow][TiCol]= A[row * k + TiNum * TILE_SZ + TiCol];
  	else
    		As[TiRow][TiCol] = 0;

 	if((TiNum * TILE_SZ + TiRow) < k && col < n)
    		Bs[TiRow][TiCol] = B[(TiNum * TILE_SZ + TiRow) * n + col];
	else
    		Bs[TiRow][TiCol] = 0;
  	__syncthreads();

        //Calculate inner product for the tile
        //Checking for matrix size to lower power and practice green computing
        if(row < m && col < n)
		for(unsigned int TiElem = 0; TiElem < TILE_SZ; TiElem++)
        		sum = sum + As[TiRow][TiElem]*Bs[TiElem][TiCol];
        __syncthreads();

        } 
    
        //Prevent writing of output to an undefined block
        if (row < m && col < n)
        	C[row * n + col] = sum;
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------
  
   //INSERT CODE HERE
    const unsigned int BLOCK_SIZE = TILE_SZ; //use 16 x 16 thread blocks
   
    dim3 block(BLOCK_SIZE, BLOCK_SIZE ,1);
    dim3 grid((n + BLOCK_SIZE - 1)/BLOCK_SIZE, (m + BLOCK_SIZE -1)/BLOCK_SIZE, 1);  
  

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE

    mysgemm<<< grid, block>>>(m, n, k, A, B, C);

}
