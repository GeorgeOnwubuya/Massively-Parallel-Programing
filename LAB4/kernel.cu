#include "hip/hip_runtime.h"
/********************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__constant__ float M_c[FILTER_SIZE][FILTER_SIZE];

/*__device__ float getElement(Matrix *N, const int row, const int col)
{
	return N->elements[row*N->width+col];
}
*/
/*__device__ void retElem(Matrix *P, const int row, const int col, float value)
{
	P->elements[row*P->width+col] = value; 

	return;
}*/

__global__ void convolution(Matrix N, Matrix P)
{
	/********************************************************************
	Determine input and output indexes of each thread
	Load a tile of the input image to shared memory
	Apply the filter on the input image tile
	Write the compute values to the output image at the correct indexes
	********************************************************************/

        //INSERT KERNEL CODE HERE
	/*int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col_zeroIndex = col - FILTER_SIZE/2;
	int row_zeroIndex = row - FILTER_SIZE/2;
	float sum = 0;

	for(int j = 0; j < FILTER_SIZE; ++j){
		for(int k = 0; k < FILTER_SIZE; ++k){
			if((row_zeroIndex + j >= 0) && (row_zeroIndex + j < N.height) &&   	   		
		   	  (col_zeroIndex + k >= 0) &&  (col_zeroIndex+ k < N.width)){		
					//sum = M_c[j][k] * getElement(&N, row_zeroIndex + j, col_zeroIndex + k);
			
					sum += M_c[j][k] * N.elements[(row_zeroIndex + j)*N.width + col_zeroIndex +k];				
			}
		}
 	}
        if( row < P.height  && col < P.width)
		
		//retElem(&P, row, col, sum);

		P.elements[row * P.width + col] = sum;*/


	int row = blockIdx.y * TILE_SIZE + threadIdx.y;
	int col = blockIdx.x * TILE_SIZE + threadIdx.x;
	int rowZeroIndex = row - FILTER_SIZE/2;
	int colZeroIndex = col - FILTER_SIZE/2;


	__shared__ float N_ds[TILE_SIZE + FILTER_SIZE - 1][TILE_SIZE + FILTER_SIZE - 1];

	if((rowZeroIndex >= 0) && (rowZeroIndex < N.height) && (colZeroIndex >= 0) && (colZeroIndex < N.width)){
		
		N_ds[threadIdx.y][threadIdx.x] = N.elements[rowZeroIndex * N.width + colZeroIndex];
	}

	else{
		N_ds[threadIdx.y][threadIdx.x] = 0.0f;
	}

	__syncthreads();
	
	float sum = 0.0f;
	
	if(threadIdx.y < TILE_SIZE && threadIdx.x < TILE_SIZE){

		for(int dr = 0; dr < FILTER_SIZE; ++dr){

			for(int dc = 0; dc < FILTER_SIZE; ++dc){

				sum += M_c[dr][dc] * N_ds[threadIdx.y + dr][threadIdx.x + dc];
			}
		}

	if(row < P.height && col < P.width){
          
        	P.elements[row * P.width + col] = sum;	
	}		
	
	}
}


