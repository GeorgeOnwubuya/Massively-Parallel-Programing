#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif



// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void preScanKernel(float *out, float *in, unsigned size, float *sum){
    // INSERT CODE HERE
	__shared__ float a_s[(2 * BLOCK_SIZE) + CONFLICT_FREE_OFFSET(2 * BLOCK_SIZE)];
	int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;

	int thid = threadIdx.x;
	thid += CONFLICT_FREE_OFFSET(thid);
	int thid_BS = threadIdx.x + BLOCK_SIZE;
	thid_BS += CONFLICT_FREE_OFFSET(thid_BS);

	a_s[thid]    = ((idx              < size)? in[idx]:            0.0f);
        a_s[thid_BS] = ((idx + BLOCK_SIZE < size)? in[idx+BLOCK_SIZE]: 0.0f);


	unsigned int ai, bi;
	unsigned int numThreads, stride;

	for(numThreads = BLOCK_SIZE, stride = 1; numThreads > 0; numThreads >>= 1, stride <<= 1){

		ai = (2 * threadIdx.x * stride + stride - 1);
		bi = (2 * threadIdx.x * stride + 2 * stride - 1);

		ai += CONFLICT_FREE_OFFSET(ai);
		bi += CONFLICT_FREE_OFFSET(bi);

	__syncthreads();

		if(threadIdx.x < numThreads)
			a_s[bi] += a_s[ai];
	}

	if(threadIdx.x == 0){
		int last_elem = 2 * BLOCK_SIZE - 1;
		last_elem += CONFLICT_FREE_OFFSET(last_elem);
		if(sum != NULL){
			sum[blockIdx.x] = a_s[last_elem];
		}
		a_s[last_elem] = 0;
	}


	for(numThreads = 1, stride = BLOCK_SIZE; numThreads <= BLOCK_SIZE; numThreads <<= 1, stride >>= 1){

		ai = (2 * threadIdx.x * stride + stride - 1);
		bi = (2 * threadIdx.x * stride + 2 * stride - 1);

		ai += CONFLICT_FREE_OFFSET(ai);
		bi += CONFLICT_FREE_OFFSET(bi);

		__syncthreads();

		if(threadIdx.x < numThreads){
		float temp = a_s[bi];
		a_s[bi] += a_s[ai];
		a_s[ai] = temp;
		}
		__syncthreads();
	}
	if(idx < size)
	out[idx] = a_s[thid];

	if(idx + BLOCK_SIZE < size)
	out[idx + BLOCK_SIZE] = a_s[thid_BS];

}


__global__ void addKernel(float *out, float *sum, unsigned size)
{
    // INSERT CODE HERE
	int idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size)
        out[idx] += sum[blockIdx.x];

        if(idx + BLOCK_SIZE < size)
        out[idx + BLOCK_SIZE] += sum[blockIdx.x];

}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void preScan(float *out, float *in, unsigned in_size)
{
	float *sum;
	unsigned num_blocks;
	hipError_t cuda_ret;
	dim3 dim_grid, dim_block;

	num_blocks = in_size/(BLOCK_SIZE*2);
	if(in_size%(BLOCK_SIZE*2) !=0) num_blocks++;

	dim_block.x = BLOCK_SIZE; dim_block.y = 1; dim_block.z = 1;
	dim_grid.x = num_blocks; dim_grid.y = 1; dim_grid.z = 1;

	if(num_blocks > 1) {
		cuda_ret = hipMalloc((void**)&sum, num_blocks*sizeof(float));
		if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

		preScanKernel<<<dim_grid, dim_block>>>(out, in, in_size, sum);
		preScan(sum, sum, num_blocks);
		addKernel<<<dim_grid, dim_block>>>(out, sum, in_size);

		hipFree(sum);
	}
	else
		preScanKernel<<<dim_grid, dim_block>>>(out, in, in_size, NULL);
}
