
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to
__global__ void histogram_kernel(unsigned int* input, unsigned int* bins, 
    unsigned int num_elements, unsigned int num_bins){

	extern __shared__ unsigned int bins_s[];
	
	//Shared Memory
	int thid = threadIdx.x;
	while(thid < num_bins){

		bins_s[thid] = 0u;
		thid += blockDim.x;
	}
	__syncthreads();


	//Histogram calculation
	unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;
	
	while(element < num_elements){

		atomicAdd(&(bins_s[input[element]]), 1);
		element += blockDim.x * gridDim.x;
	}
	__syncthreads();

	//Global Memory
	thid = threadIdx.x;
	while(thid < num_bins){

		atomicAdd(&(bins[thid]), bins_s[thid]);
		thid += blockDim.x;
	} 
}
 

__global__ void histogram_kernel_optimized(unsigned int* input, unsigned int* bins,
    unsigned int num_elements, unsigned int num_bins) {

      	// INSERT CODE HERE 
	extern __shared__ unsigned int bins_s[];
	
	//Shared memory	
	int thid = threadIdx.x;
	while ( thid < num_bins){

		bins_s[thid] = 0u;
		thid += blockDim.x; 
	}
	__syncthreads();	
 
	//Histogram calculation
	unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int accumulator = 0;
	unsigned int prev_index = 0;
	 
	while(element < num_elements){
		
		unsigned int curr_index = input[element];
		
		if(curr_index != prev_index){
				
			atomicAdd(&(bins_s[prev_index]), accumulator);
			accumulator = 1;
			prev_index = curr_index;
		
		}	
			
		else{
			accumulator++;	
		}
		element += blockDim.x * gridDim.x;
	}
	if(accumulator > 0){
		atomicAdd(&(bins_s[prev_index]), accumulator);
	}
	__syncthreads();

	//Global memory
	thid = threadIdx.x;
	while(thid < num_bins){
	
		atomicAdd(&(bins[thid]), bins_s[thid]);
		thid += blockDim.x;
	}

}

__global__ void convert_kernel(unsigned int *bins32, uint8_t *bins8,
    unsigned int num_bins) {

      // INSERT CODE HERE
	int thid = blockIdx.x * blockDim.x + threadIdx.x;

	while (thid < num_bins){
	
		//Use local  register value (avoids copying from global twice)		
		unsigned int reg_bin = bins32[thid];
		
		if(reg_bin > 255){
			bins8[thid] = 255u;
		}

		else{
			bins8[thid] = (uint8_t) reg_bin;
		}
		thid += blockDim.x * gridDim.x;
	}

}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, uint8_t* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // Create 32 bit bins
    unsigned int *bins32;
    hipMalloc((void**)&bins32, num_bins * sizeof(unsigned int));
    hipMemset(bins32, 0, num_bins * sizeof(unsigned int));

    // Launch histogram kernel using 32-bit bins
    dim3 dim_grid, dim_block;
    dim_block.x = 512; dim_block.y = dim_block.z = 1;
    dim_grid.x = 30; dim_grid.y = dim_grid.z = 1;

    //Comment out the kernel not used
    //histogram_kernel<<<dim_grid, dim_block, num_bins*sizeof(unsigned int)>>>
       // (input, bins32, num_elements, num_bins);
   histogram_kernel_optimized<<<dim_grid, dim_block, num_bins*sizeof(unsigned int)>>>
	(input, bins32, num_elements, num_bins);

    // Convert 32-bit bins into 8-bit bins
    dim_block.x = 512;
    dim_grid.x = (num_bins - 1)/dim_block.x + 1;
    convert_kernel<<<dim_grid, dim_block>>>(bins32, bins, num_bins);

    // Free allocated device memory
    hipFree(bins32);

}
